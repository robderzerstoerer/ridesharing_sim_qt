#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string>


__global__ void addKernel(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}



// Helper function for using CUDA to add vectors in parallel.
std::string addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;
	std::string output = "";

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		output = "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?";
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)& dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		output = "hipMalloc failed!";
		goto Error;
	}

	cudaStatus = hipMalloc((void**)& dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		output = "hipMalloc failed!";
		goto Error;
	}

	cudaStatus = hipMalloc((void**)& dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		output = "hipMalloc failed!";
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		output = "hipMemcpy failed!";
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		output = "hipMemcpy failed!";
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> > (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		output = "addKernel launch failed: s\n";
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		output = "hipDeviceSynchronize returned error code d after launching addKernel!\n";
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		output = "hipMemcpy failed!";
		goto Error;
	}

	output = "good";

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return output;
}



std::string cuda_main()
{
	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };

	// Add vectors in parallel.
	std::string output = addWithCuda(c, a, b, arraySize);
	if (output != "good") {
		return "addWithCuda failed!";
	}

	output = "{1,2,3,4,5} + {10,20,30,40,50} = {" + 
		std::to_string(c[0]) + "," + std::to_string(c[1]) + "," + std::to_string(c[2]) + "," + std::to_string(c[3]) + "," + std::to_string(c[4]) + "}\n";

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		return "hipDeviceReset failed!";
	}

	return output;
}